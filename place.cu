#include "hip/hip_runtime.h"
%%gpu 

#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <cstdlib>
#include <ctime>
#include <math.h>

#include<stdio.h>
#include<stdlib.h> 


using namespace std;


int n_vertex = 0;
int n_edge = 0;
vector <pair <int,int>> edges_list;

__global__ void hello(int *a, int *b, int *c, int* d) {
  printf("Hello World! \n");
}

int read_file(string filename){
    ifstream indata;
    indata.open(filename);
 
    if (!indata) {
        cerr << "Could not open the file - '"
             << filename << "'" << endl;
        return EXIT_FAILURE;
    }
 
    int a, b;
    int i = 0;
    pair <int, int> p;
 
    while (indata >> a >> b){
        if (i == 0){
          n_vertex = a;
          n_edge = b;
        }else{
            p = make_pair(a, b);
            edges_list.push_back(p);
        }
        i ++;
    }
 
    cout << n_edge << " " << n_vertex << endl;
   /* for (int i=0; i<edges_list.size(); i++){
        cout << edges_list[i].first << endl;
    }*/

    indata.close();
    return EXIT_SUCCESS;
}


int main()
{
  

    srand(time(0));
    string filename("/content/inf653_PeR_FCN/chebyshev.in");
    read_file(filename);
    
    int size_g = int(pow(n_vertex,0.5)*1.7);
    
    // STARTING THE GPU CODE
    int *adj, *loc, *gridplace, *out;
    int *d_adj, *d_gridplace, *d_out, *d_loc; // device copies of a, b, c
    int threads_per_block=0, no_of_blocks=0;
 
    // Alloc space for host copies of a, b, c and setup input values
    adj = (int *)malloc(n_vertex*n_vertex*sizeof(int)); 
    loc = (int *)malloc(2*n_vertex*sizeof(int)); 
    gridplace = (int *)malloc(size_g*size_g*sizeof(int));
    out = (int *)malloc(size_g*sizeof(int));
 
    for (int i=0; i<n_vertex; i++)  
      for (int j=0; j<n_vertex; j++)
        adj[i*n_vertex+j] = 0; //adj_matrix[i][j] = 0;
 
    int a, b;
    for (int i=0; i<edges_list.size(); i++){
      a = edges_list[i].first;
      b = edges_list[i].second;
      adj[a*n_vertex+b] = 1;
    }
 
    cout << "Print: adj matrix" << endl;
    for (int i=0; i<n_vertex; i++) { 
      for (int j=0; j<n_vertex; j++)
        cout <<  adj[i*n_vertex+j] << " ";
      cout << endl;
    }
    cout << endl;

    for (int i=0; i<size_g; i++)
      for (int j=0; j<size_g; j++)
        gridplace[i*size_g+j] = -1;

    int x,y;
   // x = rand()%(size_g);
    for (int i=0; i<n_vertex; i++){
        do{
          x = rand()%(size_g);
          y = rand()%(size_g);
        }while(gridplace[x*size_g+y] != -1);

        gridplace[x*size_g+y] = i;
        gridplace[x*size_g+y] = i;
        loc[i*2] = x;
        loc[i*2+1] = y;
     //   cout << i << " " << x << " " << y << endl;
    }
 
    cout << "Print: vertex loc in grid" << endl;
    for (int i=0; i<n_vertex; i++)
      cout << i << " " <<  loc[i*2] << " " << loc[i*2+1] << endl;
    cout << endl;
 

    //GPUT stuff
     // Alloc space for device copies of a, b, c
    hipMalloc((void **)&d_adj, n_vertex*n_vertex*sizeof(int));
    hipMalloc((void **)&d_loc, 2*n_vertex*sizeof(int));
    hipMalloc((void **)&d_gridplace, size_g*size_g*sizeof(int));
    hipMalloc((void **)&d_out, size_g*sizeof(int));

    // Copy inputs to device
    hipMemcpy(d_adj, adj, n_vertex*n_vertex*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_loc, loc, n_vertex*2*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_gridplace, gridplace, size_g*size_g*sizeof(int), hipMemcpyHostToDevice);

    threads_per_block = 512;
    no_of_blocks = size_g/threads_per_block;	
   // device_add<<<no_of_blocks,threads_per_block>>>(d_adj,d_loc,d_gridplace);
    
    hipDeviceSynchronize();
    hello<<<1,1>>>(d_adj, d_loc, d_gridplace, d_out);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(out, d_out, size_g, hipMemcpyDeviceToHost);

}
