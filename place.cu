
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <cstdlib>
#include <ctime>
#include <math.h>

#include<stdio.h>
#include<stdlib.h> 

using namespace std;

int n_vertex = 0;
int n_edge = 0;
vector <pair <int,int>> edges_list;

__global__ void print_from_gpu(void) {
	printf("Hello World! from thread [B=%d,T=%d] \
		From GPU device\n", blockIdx.x, threadIdx.x); 
}

int read_file(string filename){
    ifstream indata;
    indata.open(filename);
 
    if (!indata) {
        cerr << "Could not open the file - '"
             << filename << "'" << endl;
        return EXIT_FAILURE;
    }
 
    int a, b;
    int i = 0;
    pair <int, int> p;
 
    while (indata >> a >> b){
        if (i == 0){
          n_vertex = a;
          n_edge = b;
        }else{
            p = make_pair(a, b);
            edges_list.push_back(p);
        }
        i ++;
    }
 
    cout << n_edge << " " << n_vertex << endl;
   /* for (int i=0; i<edges_list.size(); i++){
        cout << edges_list[i].first << endl;
    }*/

    indata.close();
    return EXIT_SUCCESS;
}

int main()
{
    srand(time(0));
    string filename("chebyshev.in");
    read_file(filename);

    int vertex_loc[n_vertex][2];

    int adj_matrix[n_vertex][n_vertex];
    for (int i=0; i<n_vertex; i++)  
      for (int j=0; j<n_vertex; j++)
        adj_matrix[i][j] = -1;
 
    for (int i=0; i<edges_list.size(); i++){
      adj_matrix[edges_list[i].first][edges_list[i].second] = 1;
      //cout << "aaa"<< edges_list[i].first << " " << edges_list[i].second << endl;
    }
 
    cout << "Print: adj matrix" << endl;
    for (int i=0; i<n_vertex; i++) { 
      for (int j=0; j<n_vertex; j++)
        cout << adj_matrix[i][j] << " ";
      cout << endl;
    }
    cout << endl;

    int size_g = int(pow(n_vertex,0.5)*1.7);
    int grid[size_g][size_g];
 
    for (int i=0; i<size_g; i++)
      for (int j=0; j<size_g; j++)
        grid[i][j] = 0;

    int x,y;
    for (int i=0; i<n_vertex; i++){
        do{
          x = rand()%(size_g);
          y = rand()%(size_g);
        }while(grid[x][y] != 0);

        grid[x][y] = i;
        vertex_loc[i][0] = x; 
        vertex_loc[i][1] = y;
     //   cout << i << " " << x << " " << y << endl;
    }
 
    cout << "Print: vertex loc in grid" << endl;
    for (int i=0; i<n_vertex; i++)
      cout << i << " " <<  vertex_loc[i][0] << " " << vertex_loc[i][1] << endl;
    cout << endl;
 
    cout << "Print: placement grid" << endl;
    for (int i=0; i<size_g; i++){
     for (int j=0; j<size_g; j++)
      cout << grid[i][j] << " ";
      cout << endl;
    }
 
    //Sai de i e vai pra j
    int origin_x, origin_y;
    int dest_x, dest_y;
    int total = 0;
    for (int i=0; i<n_vertex; i++) { 
      for (int j=0; j<n_vertex; j++)
        if (adj_matrix[i][j] == 1){ // tem aresta ligando
            origin_x = vertex_loc[i][0];
            origin_y = vertex_loc[i][1];
            dest_x = vertex_loc[j][0];
            dest_y = vertex_loc[j][1];
            total += abs(origin_x - dest_x) + abs(origin_y - dest_y); 
            //cout << cost << endl;
        } 

    }
    
    print_from_gpu<<<1,2>>>();
    hipDeviceSynchronize();
       
}
